// gpu_processing.cu
#include "helper.h"    
#include "gpu_processing.cuh"
#include "kernel.cuh"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <numeric>
#include <iostream>

#define CUDA_CHECK(err) \
    if(err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " at line " << __LINE__ << std::endl; exit(EXIT_FAILURE); }

GpuProcessor::GpuProcessor(const cv::Mat& hostImage) {
    CV_Assert(hostImage.type() == CV_64F && hostImage.channels() == 1);
    width = hostImage.cols;
    height = hostImage.rows;
    N = static_cast<size_t>(width) * height;
    allocateDevice(hostImage);
    computeThresholds(hostImage);
}

GpuProcessor::~GpuProcessor() {
    freeDevice();
}

void GpuProcessor::allocateDevice(const cv::Mat& hostImage) {
    size_t bufBytes = N * sizeof(double);
    size_t grayBytes = N * sizeof(unsigned char);
    size_t colBytes = N * 3 * sizeof(unsigned char);

    CUDA_CHECK(hipMalloc(&d_buf, bufBytes));
    CUDA_CHECK(hipMalloc(&d_lin, grayBytes));
    CUDA_CHECK(hipMalloc(&d_log, grayBytes));
    CUDA_CHECK(hipMalloc(&d_blur, grayBytes));
    CUDA_CHECK(hipMalloc(&d_col_lin, colBytes));
    CUDA_CHECK(hipMalloc(&d_col_log, colBytes));

    CUDA_CHECK(hipMemcpy(d_buf, hostImage.ptr<double>(), bufBytes,
        hipMemcpyHostToDevice));
}

void GpuProcessor::freeDevice() {
    hipFree(d_buf);
    hipFree(d_lin);
    hipFree(d_log);
    hipFree(d_blur);
    hipFree(d_col_lin);
    hipFree(d_col_log);
}

void GpuProcessor::computeThresholds(const cv::Mat& hostImage) {
    th = ::computeThresholds(hostImage);
}

std::pair<float, float> GpuProcessor::benchmarkLinear(int runs) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    std::vector<float> times(runs);
    int threads = 256;
    int blocks = static_cast<int>((N + threads - 1) / threads);

    for (int i = 0; i < runs; ++i) {
        CUDA_CHECK(hipEventRecord(start));
        gpu_linear_percentile_stretch << <blocks, threads >> > (d_buf, d_lin,
            N, th.v_low, th.v_high);
        gpu_gaussian_blur << <blocks, threads >> > (d_lin, d_blur, N, width, height);
        gpu_colorize << <blocks, threads >> > (d_blur, d_col_lin, N);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&times[i], start, stop));
    }
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    float min_t = *std::min_element(times.begin(), times.end());
    float avg_t = std::accumulate(times.begin(), times.end(), 0.0f) / runs;
    return { min_t / 1000.0f, avg_t / 1000.0f };
}

std::pair<float, float> GpuProcessor::benchmarkLog(int runs) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    std::vector<float> times(runs);
    int threads = 256;
    int blocks = static_cast<int>((N + threads - 1) / threads);

    for (int i = 0; i < runs; ++i) {
        CUDA_CHECK(hipEventRecord(start));
        gpu_log_scale_stretch << <blocks, threads >> > (d_buf, d_log,
            N, th.minV, th.range);
        gpu_gaussian_blur << <blocks, threads >> > (d_log, d_blur, N, width, height);
        gpu_colorize << <blocks, threads >> > (d_blur, d_col_log, N);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&times[i], start, stop));
    }
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    float min_t = *std::min_element(times.begin(), times.end());
    float avg_t = std::accumulate(times.begin(), times.end(), 0.0f) / runs;
    return { min_t / 1000.0f, avg_t / 1000.0f };
}

cv::Mat GpuProcessor::getLinearColor() {
    cv::Mat img(height, width, CV_8UC3);
    CUDA_CHECK(hipMemcpy(img.data, d_col_lin,
        N * 3 * sizeof(unsigned char),
        hipMemcpyDeviceToHost));
    return img;
}

cv::Mat GpuProcessor::getLogColor() {
    cv::Mat img(height, width, CV_8UC3);
    CUDA_CHECK(hipMemcpy(img.data, d_col_log,
        N * 3 * sizeof(unsigned char),
        hipMemcpyDeviceToHost));
    return img;
}
